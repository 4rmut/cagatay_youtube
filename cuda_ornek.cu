
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <cstdlib>

// Kolaylık olsun
using namespace std;

// Tipik bir C++ fonksiyonu 
void carp(int n, float *x, float *y, float *z)
{
  for (int i = 0; i < n; i++)
  {
      z[i] = x[i] * y[i];
  }   
}


// Ustteki fonksiyonun CUDA versiyonu
__global__
void carp_cuda(int n, float *x, float *y, float *z)
{
  for (int i = threadIdx.x; i < n; i += blockDim.x)
  {
      z[i] = x[i] * y[i];
  }       
}

int main(int argc, char *argv[])
{
  // Çok büyük bir sayı belirleyelim
  int N = 10000;

  float *x_gpu, *y_gpu, *z_gpu, *x_cpu, *y_cpu, *z_cpu;

  // GPU ve CPU tarafindan ulasilabilen memory ayirtalim
  hipMallocManaged(&x_gpu, N * sizeof(float));
  hipMallocManaged(&y_gpu, N * sizeof(float));
  hipMallocManaged(&z_gpu, N * sizeof(float));

  // Sadece CPU tarafindan ulasilabilen memory ayirtalim
  x_cpu = new float[N];
  y_cpu = new float[N];
  z_cpu = new float[N];

  // 
  for (int i = 0; i < N; ++i) {
    x_gpu[i] = 1.0f;
    y_gpu[i] = 2.0f;
    x_cpu[i] = 1.0f;
    y_cpu[i] = 2.0f;
  }

  // Fonksiyonu GPU'da argv[1] blokta ve her blokta argv[2] thread
  // olacak sekilde çagiralım
  int blok_sayisi = atoi(argv[1]);
  int thread_sayisi = atoi(argv[2]);

  carp_cuda<<<blok_sayisi, thread_sayisi>>>(N, x_gpu, y_gpu, z_gpu);

  // GPU'yu bekleyelim de isini bitirsin, yoksa ortam karisir.
  hipDeviceSynchronize();

  // Normal CPU fonlsiyonunu çagiralım
  carp(N, x_cpu, y_cpu, z_cpu);

  // Bakalim dogru mu yaptik?
  // z_gpu ve z_cpu ayni degerlere sahip olması lazim 
  for(int i = 0; i < N; ++i) 
      cout << z_cpu[i] << " " << z_gpu[i] << endl;

  // Release the Kraken - Kraken'i saliverin gelsin. 
  hipFree(x_gpu);
  hipFree(y_gpu);
  hipFree(z_gpu);
  delete [] x_cpu;
  delete [] y_cpu;
  delete [] z_cpu;
  
  return 0;
}
