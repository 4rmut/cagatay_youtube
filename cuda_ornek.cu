
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <cstdlib>

// Kolayl�k olsun
using namespace std;

// Tipik bir C++ fonksiyonu 
void carp(int n, float *x, float *y, float *z)
{
  for (int i = 0; i < n; i++)
  {
      z[i] = x[i] * y[i];
  }   
}


// �stteki fonksiyonun CUDA versiyonu
__global__
void carp_cuda(int n, float *x, float *y, float *z)
{
  for (int i = threadIdx.x; i < n; i += blockDim.x)
  {
      z[i] = x[i] * y[i];
  }       
}

int main(int argc, char *argv[])
{
  // �ok b�y�k bir say� belirleyelim
  int N = 10000;

  float *x_gpu, *y_gpu, *z_gpu, *x_cpu, *y_cpu, *z_cpu;

  // GPU ve CPU taraf�ndan ula�al�bilen memory ay�rtal�m
  hipMallocManaged(&x_gpu, N * sizeof(float));
  hipMallocManaged(&y_gpu, N * sizeof(float));
  hipMallocManaged(&z_gpu, N * sizeof(float));

  // Sadece CPU taraf�ndan ula��labilen memory ay�rtal�m
  x_cpu = new float[N];
  y_cpu = new float[N];
  z_cpu = new float[N];

  // 
  for (int i = 0; i < N; ++i) {
    x_gpu[i] = 1.0f;
    y_gpu[i] = 2.0f;
    x_cpu[i] = 1.0f;
    y_cpu[i] = 2.0f;
  }

  // Fonksiyonu GPU'da argv[1] blokta ve her blokta argv[2] thread
  // olacak �ekilde �a��ral�m
  int blok_sayisi = atoi(argv[1]);
  int thread_sayisi = atoi(argv[2]);

  carp_cuda<<<blok_sayisi, thread_sayisi>>>(N, x_gpu, y_gpu, z_gpu);

  // GPU'yu bekleyelim de i�ini bitirsin, yoksa ortam kar���r.
  hipDeviceSynchronize();

  // Normal CPU fonsiyonunu �a��ral�m
  carp(N, x_cpu, y_cpu, z_cpu);

  // Bakal�m do�ru mu yapt�k?
  // z_gpu ve z_cpu ayn� de�erlere sahip olmas� laz�m 
  for(int i = 0; i < N; ++i) 
      cout << z_cpu[i] << " " << z_gpu[i] << endl;

  // Release the Kraken - Kraken'� sal�verin gelsin. 
  hipFree(x_gpu);
  hipFree(y_gpu);
  hipFree(z_gpu);
  delete [] x_cpu;
  delete [] y_cpu;
  delete [] z_cpu;
  
  return 0;
}

